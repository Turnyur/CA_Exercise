#include "hip/hip_runtime.h"
#define _POSIX_C_SOURCE 199309L
#include <stdint.h>
#include <stdio.h>
#include <inttypes.h>
#include "jacobi.h"

// Uncomment the following line to enable twofold unrolling

__global__ void update_grid(double *grid_source, double *grid_target, uint32_t x, uint32_t y)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    //if (i < x && j < x)
    if (i > 0 && i < x && j > 0 && j < x)
    {
        grid_target[x * i + j] = 0.25 * (grid_source[x * i + (j - 1)] + grid_source[x * i + (j + 1)] +
                                         grid_source[x * (i - 1) + j] + grid_source[x * (i + 1) + j]);
    }

    //printf("i:%" PRId32  " j:%" PRId32 "\n", i,j);
}

void serial_update_grid(double *grid_source, double *grid_target, uint32_t x, uint32_t y)
{
    for (uint32_t i = 1; i < y - 1; ++i)
    {
        for (uint32_t j = 1; j < x - 1; ++j)
        {
            grid_target[x * i + j] = 0.25 * (grid_source[x * i + (j - 1)] + grid_source[x * i + (j + 1)] +
                                             grid_source[x * (i - 1) + j] + grid_source[x * (i + 1) + j]);
        }
    }
}

__global__ void addKernel(double *grid_source, double *grid_target, double *result, uint32_t x, uint32_t y)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < x && j < x)
    {
        result[x * i + j] = grid_source[x * i + j ] + grid_target[x * i +  j];
    }
}
