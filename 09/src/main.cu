#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#include <time.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C"
{
#include "draw.h"
}
extern "C"
{
#include "get_time.h"
}
extern "C"
{
#include "helpers.h"
}
#include "jacobi.h"

// #define GRID_SIZE 3000000000

int main(int argc, char *argv[])
{
	uint64_t start = 0u;
	uint64_t stop = 0u;
	uint64_t actual_runtime_without_copy = 0u;
	uint64_t actual_runtime_with_copy = 0u;

	//uint64_t actual_runtime = 0u;
	uint64_t runs = 0u;
	//uint64_t minimal_runtime = 1000000; // 1 sec
	//uint64_t minimal_runtime = 100000; // 100 millisec
	uint64_t minimal_runtime = 10000000; //10 sec

	double *d_gridSource;
	double *d_gridTarget;
	// double *d_result;

	uint64_t gridLength = GRID_SIZE / sizeof(double);
	uint64_t edge_size = sqrt(gridLength) - 4;
	//printf(" edge_size:%" PRId64 "\n", edge_size);

	uint64_t num_rows = edge_size;
	uint64_t num_cols = num_rows;

	double *h_gridSource = (double *)malloc(gridLength * sizeof(double));
	double *h_gridTarget = (double *)malloc(gridLength * sizeof(double));
	double *h_resultGrid = (double *)malloc(gridLength * sizeof(double));

	initialize_grid(h_gridSource, num_rows, num_cols);
	initialize_grid(h_gridTarget, num_rows, num_cols);

	// initializeAddKernel(h_gridSource, num_rows, num_cols);
	// initializeAddKernel(h_gridTarget, num_rows, num_cols);

	printf("\nJOB_INFO:\n \tROWS:%" PRId64 " COLUMNS:%" PRId64 "\n", num_rows, num_cols);
	printf(" \tNUMBER OF GRID POINTS:%" PRId64, gridLength);

	// Allocate memory
	hipMalloc((void **)&d_gridSource, sizeof(double) * gridLength);
	hipMalloc((void **)&d_gridTarget, sizeof(double) * gridLength);
	// hipMalloc((void **)&d_result, sizeof(double) * gridLength);

	// Copy data from host to device
	uint64_t preMemTimeStart = get_time_us();
	hipMemcpy(d_gridSource, h_gridSource, sizeof(double) * gridLength, hipMemcpyHostToDevice);
	hipMemcpy(d_gridTarget, h_gridTarget, sizeof(double) * gridLength, hipMemcpyHostToDevice);
	uint64_t preMemTime = get_time_us() - preMemTimeStart;
	/*if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed for one of the memory copy operations: %s\n", hipGetErrorString(cudaStatus));
	}*/

	// set kernel dimension
	// RTX GPU SM
	// 32x32 =1024
	uint RTX_MAX_THREAD_BLOCK = 1024;
	uint x_count = sqrt(RTX_MAX_THREAD_BLOCK);
	uint y_count = x_count;

	dim3 block_size(x_count, y_count);
	uint x = (num_cols + block_size.x - 1) / block_size.x;
	uint y = (num_rows + block_size.y - 1) / block_size.y;
	dim3 grid_size(x, y);

	printf("\n\n \tGPU KERNEL:\n \t\tGRID_SIZE:%dx%d  BLOCK_SIZE:%dx%d  \n",  grid_size.x, grid_size.y, block_size.y, block_size.x);

	

	for (runs = 1u; actual_runtime_without_copy < minimal_runtime; runs = runs << 1u)
	{
		start = get_time_us();

		for (uint64_t i = 0u; i < runs; i++)
		{
			// serial_update_grid(h_gridSource, h_gridTarget, num_cols, num_rows);
			// swap(&h_gridSource, &h_gridTarget);

			update_grid<<<grid_size, block_size>>>(d_gridSource, d_gridTarget, num_cols, num_rows);
			// addKernel<<<grid_size, block_size>>>(d_gridSource, d_gridTarget, d_result, num_cols, num_rows);

			hipDeviceSynchronize();
			swap(&d_gridTarget, &d_gridSource);
		}

		stop = get_time_us();
		actual_runtime_without_copy = stop - start;
	}

	uint64_t postMemTimeStart = get_time_us();
	hipMemcpy(h_resultGrid, d_gridTarget, sizeof(double) * gridLength, hipMemcpyDeviceToHost);
	uint64_t postMemTime = get_time_us() - preMemTimeStart;

	actual_runtime_with_copy = actual_runtime_without_copy + (preMemTime + postMemTime);

	// debugger:
	// h_resultGrid = h_gridTarget; //to be deleted

	// printGrid(h_resultGrid, num_rows, num_cols);
	printf("\n\nResult Validity:");
	printf("\n \th_resultGrid[0]: %f", h_resultGrid[0]);
	printf("\n \th_resultGrid[1]: %f\n", h_resultGrid[1]);
	int numDots = 3;
	for (int i = 0; i < numDots; i++)
	{
		printf("\t  .\n");
	}
	uint offset = num_cols * num_rows;
	printf(" \th_resultGrid[%d]: %f", offset - 2, h_resultGrid[offset - 2]);
	printf("\n \th_resultGrid[%d]: %f\n", offset - 1, h_resultGrid[offset - 1]);

	char file_path[50];
	sprintf(file_path, "jacobi_heat_map_%" PRId64 "x%" PRId64 ".ppm", num_rows, num_cols);

	// draw_grid(h_resultGrid, num_cols, num_rows, file_path);

	// Calculate the Bandwidth
	runs= runs/2;
	uint64_t total_bytes = (5 * sizeof(double) * gridLength * runs); // =>4 load operations, 1 store operation
	uint64_t mega_bytes_per_second = (total_bytes / (actual_runtime_without_copy * 1e-6)) / 1e6;

	printf("\n\nTotal Bytes Loaded: %d GiB\n\n", (int)(total_bytes / 1e9));
	// Print Benchmark Output
	printf("\nFunction\t\t MB/s\t\t Time\n\n");
	printf("JACOBI(Without COPY)   \t %" PRId64 "    \t %" PRId64 "\n", mega_bytes_per_second, actual_runtime_without_copy);

	total_bytes = ((5 * sizeof(double)*runs) + sizeof(double)) * gridLength;
	mega_bytes_per_second = (total_bytes / (actual_runtime_with_copy * 1e-6)) / 1e6;
	fprintf(stdout, "JACOBI(With COPY)   \t %" PRId64 "    \t %" PRId64 "\n\n\n", mega_bytes_per_second, actual_runtime_with_copy);

	free(h_gridSource);
	free(h_gridTarget);
	free(h_resultGrid);

	hipFree(d_gridSource);
	hipFree(d_gridTarget);
	// hipFree(d_result);

	return 0;
}
