//#define _POSIX_C_SOURCE 199309L
#include <stdint.h>
#include <stdio.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

extern "C"
{
#include "dgemm.h"
}

// Uncomment the following line to enable twofold unrolling

void gpu_blas_mmul(double *A, double *B, double *C, uint64_t m, uint64_t k, uint64_t n)
{
    int lda = m, ldb = k, ldc = m;
    double alf = 1;
    double bet = 0;
    double *alpha = &alf;
    double *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);
}