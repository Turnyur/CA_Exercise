#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

extern "C"
{
#include "get_time.h"
}
extern "C"
{
#include "dgemm.h"
}


int main(int argc, char *argv[])
{
	uint64_t start = 0u;
	uint64_t stop = 0u;
	uint64_t actual_runtime_without_copy = 0u;
	uint64_t actual_runtime_with_copy = 0u;

	// uint64_t actual_runtime = 0u;
	uint64_t runs = 0u;
	//uint64_t minimal_runtime = 1000000;

	uint64_t ArrayLength = ARRAY_SIZE / sizeof(double);
	uint64_t MATRIX_ROW_LENGTH = sqrt(ArrayLength / 3); // Share among square arrays
	uint64_t MATRIX_COL_LENGTH = MATRIX_ROW_LENGTH;

	uint64_t N = MATRIX_ROW_LENGTH;

	double *h_A = (double *)malloc(MATRIX_ROW_LENGTH * MATRIX_COL_LENGTH * sizeof(double));
	double *h_C = (double *)malloc(MATRIX_ROW_LENGTH * MATRIX_COL_LENGTH * sizeof(double));
	double *h_B = (double *)malloc(MATRIX_ROW_LENGTH * MATRIX_COL_LENGTH * sizeof(double));

	printf("\nJOB_INFO:\n \tROWS:%" PRId64 " COLUMNS:%" PRId64 "\n", N, N);
	//printf(" \tNUMBER OF GRID POINTS:%ld" , ARRAY_SIZE);

	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A, MATRIX_ROW_LENGTH * MATRIX_COL_LENGTH * sizeof(double));
	hipMalloc(&d_B, MATRIX_ROW_LENGTH * MATRIX_COL_LENGTH * sizeof(double));
	hipMalloc(&d_C, MATRIX_ROW_LENGTH * MATRIX_COL_LENGTH * sizeof(double));

	for (uint64_t i = 0; i < N; ++i)
	{
		for (uint64_t j = 0; j < N; ++j)
		{
			h_A[N * i + j] = 2;
			h_B[N * i + j] = 3;
		}
	}

	// Copy data from host to device
	uint64_t preMemTimeStart = get_time_us();
	hipMemcpy(d_A, h_A, sizeof(double) * (N * N), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(double) * (N * N), hipMemcpyHostToDevice);
	uint64_t preMemTime = get_time_us() - preMemTimeStart;
	/*if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed for one of the memory copy operations: %s\n", hipGetErrorString(cudaStatus));
	}*/

	// set kernel dimension
	// RTX GPU SM
	// 32x32 =1024
	//uint RTX_MAX_THREAD_BLOCK = 1024;
	//uint x_count = sqrt(RTX_MAX_THREAD_BLOCK);
	//uint y_count = x_count;

	//dim3 block_size(x_count, y_count);
	//uint x = (MATRIX_COL_LENGTH + block_size.x - 1) / block_size.x;
	//uint y = (MATRIX_ROW_LENGTH + block_size.y - 1) / block_size.y;
	//dim3 grid_size(x, y);

	//printf("\n\n \tGPU KERNEL:\n \t\tGRID_SIZE:%dx%d  BLOCK_SIZE:%dx%d  \n", grid_size.x, grid_size.y, block_size.y, block_size.x);

	// for (runs = 1u; actual_runtime_without_copy < minimal_runtime; runs = runs << 1u)
	//{
	start = get_time_us();

	// for (uint64_t i = 0u; i < runs; i++)
	//{
	gpu_blas_mmul(d_A, d_B, d_C, MATRIX_ROW_LENGTH, MATRIX_COL_LENGTH, MATRIX_COL_LENGTH);
	//}

	stop = get_time_us();
	actual_runtime_without_copy = stop - start;
	//}

	uint64_t postMemTimeStart = get_time_us();
	hipMemcpy(h_C, d_C, sizeof(double) * (N * N), hipMemcpyDeviceToHost);
	uint64_t postMemTime = get_time_us() - preMemTimeStart;

	actual_runtime_with_copy = actual_runtime_without_copy + (preMemTime + postMemTime);

	printf("\n\nResult Validity:");
	printf("\n \th_C[0]: %f", h_C[0]);
	printf("\n \th_C[1]: %f\n", h_C[1]);
	int numDots = 3;
	for (int i = 0; i < numDots; i++)
	{
		printf("\t  .\n");
	}
	uint offset = N * N;
	printf(" \th_C[%d]: %f", offset - 2, h_C[offset - 2]);
	printf("\n \th_C[%d]: %f\n", offset - 1, h_C[offset - 1]);

	// Calculate the Bandwidth
	//runs = runs / 2;

	uint64_t dgemm_FLOPS = 2 * pow(N, 3);

	uint64_t giga_flops_per_second = (dgemm_FLOPS / (actual_runtime_without_copy * 1e-6)) / 1e9;

	// Print Benchmark Output
	printf("\n\nFunction\t\t GFLOP/s\t\t Time\n\n");
	printf("CBLAS(Without COPY)   \t %" PRId64 "    \t %" PRId64 "\n", giga_flops_per_second, actual_runtime_without_copy);

	giga_flops_per_second = (dgemm_FLOPS / (actual_runtime_with_copy * 1e-6)) / 1e9;
	fprintf(stdout, "CBLAS(With COPY)   \t %" PRId64 "    \t %" PRId64 "\n\n\n", giga_flops_per_second, actual_runtime_with_copy);



	free(h_A);
	free(h_B);
	free(h_C);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
