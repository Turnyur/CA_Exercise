#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#include <time.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "get_time.h"
#include "vec_triad.h"

__global__ void triad_kernel(float *A, float *B, float *C, float c, uint32_t N)
{
	uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
		A[i] = B[i] * c + C[i];
}

uint64_t get_time_us(void)
{
	struct timespec a;
	clock_gettime(CLOCK_MONOTONIC, &a);
	return (uint64_t)(((double)a.tv_nsec / 1000.0) + ((double)a.tv_sec * (1000.0 * 1000.0)));
}

void serial_vec_triad(float *A, float *B, float *C, float c, uint64_t vector_size)
{

	for (uint32_t k = 0; k < vector_size; ++k)
	{
		A[k] = B[k] * c + C[k];
	}
	printf("serial_vec_triad:vector_size %" PRId64 "\n", vector_size);
	printf("\n A_RESULT:0: %f", A[0]);
	printf("\n A_RESULT:LAST %f\n", A[vector_size - 1]);
}

int main(int argc, char *argv[])
{
	uint64_t start = 0u;
	uint64_t stop = 0u;
	uint64_t actual_runtime_without_copy = 0u;
	uint64_t actual_runtime_with_copy = 0u;
	float *d_A;
	float *d_B;
	float *d_C;

	uint32_t ARRAY_LENGHT = STREAM_ARRAY_SIZE / sizeof(float);
	// Initialize Arrays
	float *h_A = (float *)malloc(ARRAY_LENGHT * sizeof(float));
	float *h_B = (float *)malloc(ARRAY_LENGHT * sizeof(float));
	float *h_C = (float *)malloc(ARRAY_LENGHT * sizeof(float));
	float c = 0.3; // 0.0 - 1.0

	printf("\n\nSTREAM_ARRAY_SIZE: %d\n", STREAM_ARRAY_SIZE);
	printf("ARRAY_LENGTH: %d\n", ARRAY_LENGHT);

	for (uint32_t i = 0; i < ARRAY_LENGHT; i++)
	{
		h_B[i] = 3.0;
		h_C[i] = 2.0;
	}
	// Allocate memory
	hipError_t cudaStatus = hipMalloc((void **)&d_A, sizeof(float) * ARRAY_LENGHT);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc for d_A failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipMalloc((void **)&d_B, sizeof(float) * ARRAY_LENGHT);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc for d_B failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipMalloc((void **)&d_C, sizeof(float) * ARRAY_LENGHT);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc for d_C failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// Copy data from host to device
	cudaStatus = hipMemcpy(d_B, h_B, sizeof(float) * ARRAY_LENGHT, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy for d_B failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMemcpy(d_C, h_C, sizeof(float) * ARRAY_LENGHT, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy for d_C failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// set kernel dimension
	// RTX GPU SM

	uint RTX_MAX_THREAD_BLOCK = 1024;
	int numBlocks = (ARRAY_LENGHT + RTX_MAX_THREAD_BLOCK) / RTX_MAX_THREAD_BLOCK;

	dim3 grid_size(numBlocks);
	dim3 block_size(RTX_MAX_THREAD_BLOCK);

	// Benchmark Triad
	start = get_time_us();
	triad_kernel<<<grid_size, block_size>>>(d_A, d_B, d_C, c, ARRAY_LENGHT);
	hipDeviceSynchronize();

	// Measuring without Copy operation
	stop = get_time_us();
	actual_runtime_without_copy = stop - start;

	// Copy results back to host
	cudaStatus = hipMemcpy(h_A, d_A, sizeof(float) * ARRAY_LENGHT, hipMemcpyDeviceToHost);
	/*if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy for d_A failed: %s\n", hipGetErrorString(cudaStatus));
	}*/

	// With Copy operation
	stop = get_time_us();
	actual_runtime_with_copy = stop - start;

	// Used for debugging
	printf("\n\nResult Validity:");
	printf("\n \tA[0]: %f", h_A[0]);
	printf("\n \tA[1]: %f\n", h_A[1]);
	int numDots =3;
	for (int i = 0; i < numDots; i++)
	{
		printf("\t  .\n");
	}
	printf(" \tA[%d]: %f", ARRAY_LENGHT - 2, h_A[ARRAY_LENGHT - 2]);
	printf("\n \tA[%d]: %f", ARRAY_LENGHT - 1, h_A[ARRAY_LENGHT - 1]);

	// Calculate the Bandwidth
	uint64_t total_bytes = (3 * sizeof(float) * ARRAY_LENGHT); // =>2 load operations, 1 store operation
	uint64_t mega_bytes_per_second = (total_bytes / (actual_runtime_without_copy * 1e-6)) / 1e6;

	printf("\n\nTotal Bytes Loaded: %d GiB\n\n", (int)(total_bytes / 1e9));
	// Print Benchmark Output
	printf("\nFunction\t\t MB/s\t\t Time\n\n");
	fprintf(stdout, "Triad(Without COPY)   \t %" PRId64 "    \t %" PRId64 "\n", mega_bytes_per_second, actual_runtime_without_copy);


	total_bytes = (3 * sizeof(float) + sizeof(float)) * ARRAY_LENGHT;
	mega_bytes_per_second = (total_bytes / (actual_runtime_with_copy * 1e-6)) / 1e6;
	fprintf(stdout, "Triad(With COPY)   \t %" PRId64 "    \t %" PRId64 "\n\n\n", mega_bytes_per_second, actual_runtime_with_copy);

	free(h_A);
	free(h_B);
	free(h_C);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
